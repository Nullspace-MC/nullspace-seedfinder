
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#define DEFAULT_START_SEED 0
#define DEFAULT_END_SEED 281474976710656
#define DEFAULT_QUADHUT_LIST "./seeds/quadhut_bases.txt"

#define cudaCheckErrors(msg) \
    do { \
	hipError_t __err = hipGetLastError(); \
	if(__err != hipSuccess) { \
	    fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
		msg, hipGetErrorString(__err), \
		__FILE__, __LINE__); \
	    exit(1);\
	} \
    } while(0)

/* queues the given seed in a buffer for a host thread to write the seed to
 * the output file
 */
__device__ void queueForOutput(int64_t seed, volatile int64_t *buf,
		    int thread_id) {
    while(buf[thread_id] != -1);
    buf[thread_id] = seed;
}

/* returns the square of the distance between two points */
__device__ int distanceSquared(int2 *p1, int2 *p2) {
    const int dx = p2->x - p1->x;
    const int dy = p2->y - p1->y;
    return (dx * dx) + (dy * dy);
}

/* returns the coordinates of the structure of a given region */
__device__ int2 getStructurePos(int64_t seed, int regX, int regZ) {
    // set seed
    seed = regX * 341873128712 + regZ * 132897987541 + seed + 14357617;
    seed = (seed ^ 0x5deece66dLL);

    // get chunk x within region
    seed = (seed * 0x5deece66dLL + 0xbLL) & 0xffffffffffff;
    int x = (int)(seed >> 17) % 24;

    // get chunk z within region
    seed = (seed * 0x5deece66dLL + 0xbLL) & 0xffffffffffff;
    int z = (int)(seed >> 17) % 24;

    // get block position from chunk position
    x = ((regX * 32 + x) << 4) + 9;
    z = ((regZ * 32 + z) << 4) + 9;
    return make_int2(x, z);
}

/* searches the base seed space for quad structures */
__global__ void findOriginStructures(int64_t start, int64_t range,
		    volatile int64_t *buf) {
    const int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int num_threads = gridDim.x * blockDim.x;
    
    const int64_t thread_start = start + (thread_id * range / num_threads);
    const int64_t thread_end = start + ((thread_id + 1) * range / num_threads);

    for(int64_t seed = thread_start; seed < thread_end; ++seed) {
	// get hut chunks
	int2 huts[4];
	huts[0] = getStructurePos(seed, 0, 0);
	huts[1] = getStructurePos(seed, 0, 1);
	huts[2] = getStructurePos(seed, 1, 0);
	huts[3] = getStructurePos(seed, 1, 1);

	// quadhut check
	int2 center = make_int2(
	    (huts[0].x + huts[1].x + huts[2].x + huts[3].x) / 4,
	    (huts[0].y + huts[1].y + huts[2].y + huts[3].y) / 4
	);
	int isQuadhut = 1;
	for(int i = 0; i < 4; ++i) {
	    if(distanceSquared(&huts[i], &center) > 16384) {
		isQuadhut = 0;
		break;
	    }
	}

	if(isQuadhut) {
	    queueForOutput(seed, buf, thread_id);
	}
    }
}

void usage() {
    fprintf(stderr, "USAGE:\n");
    fprintf(stderr, "  find_origin_structures [OPTION]...\n");
    fprintf(stderr, "    --help    (-h)\n");
    fprintf(stderr, "    --start_seed=<integer>\n");
    fprintf(stderr, "        (Defaults to 0)\n");
    fprintf(stderr, "    --end_seed=<integer>\n");
    fprintf(stderr, "        (Defaults to 281474976710656)\n");
    fprintf(stderr, "    --work_unit=<integer>\n");
    fprintf(stderr, "        (Optional, specifies a work unit 0-255)\n");
    fprintf(stderr, "    --num_units=<integer>\n");
    fprintf(stderr, "        (Defaults to 1)\n");
    fprintf(stderr, "    --quadhut_list=<file path>\n");
    fprintf(stderr, "        (Defaults to ./seeds/quadhut_list.txt)\n");
}

int main(int argc, char *argv[]) {
    int64_t start_seed = DEFAULT_START_SEED;
    int64_t end_seed = DEFAULT_END_SEED;
    char *quadhut_list_filename = DEFAULT_QUADHUT_LIST;
    char *file_access_mode = "a";
    
    int work_unit = -1;
    int num_units = 1;

    // parse args
    char *endptr;
    for(int a = 1; a < argc; ++a) {
	if(!strncmp(argv[a], "--start_seed=", 13)) {
	    start_seed = strtoll(argv[a] + 13, &endptr, 0);
	} else if(!strncmp(argv[a], "--end_seed=", 11)) {
	    end_seed = strtoll(argv[a] + 11, &endptr, 0);
	} else if(!strncmp(argv[a], "--work_unit=", 12)) {
	    work_unit = (int)strtoll(argv[a] + 12, &endptr, 0);
	} else if(!strncmp(argv[a], "--num_units=", 12)) {
	    num_units = (int)strtoll(argv[a] + 12, &endptr, 0);
	} else if(!strncmp(argv[a], "--quadhut_list=", 15)) {
	    quadhut_list_filename = argv[a] + 15;
	} else if(!strcmp(argv[a], "--help") || !strcmp(argv[a], "-h")) {
	    usage();
	    exit(0);
	} else {
	    fprintf(stderr, "Unrecognized argument: %s\n", argv[a]);
	    usage();
	    exit(-1);
	}
    }

    if(work_unit >= 0 && work_unit < 256) {
	start_seed = 1099511627776*((int64_t)work_unit);
	end_seed = start_seed + 1099511627776*((int64_t)num_units);
    }
    int64_t seed_range = end_seed - start_seed;

    FILE *quadhut_list = fopen(quadhut_list_filename, file_access_mode);

    if(quadhut_list == NULL) {
	fprintf(stderr, "Could not open \"%s\"\n", quadhut_list_filename);
	exit(-1);
    }

    int grid_size, block_size, thread_cnt;
    hipDeviceGetAttribute(&grid_size, hipDeviceAttributeMultiprocessorCount, 0);
    block_size = 1024;
    cudaCheckErrors("hipDeviceGetAttribute fail");
    thread_cnt = grid_size * block_size;

    // buffer setup
    hipSetDeviceFlags(hipDeviceMapHost);
    volatile int64_t *h_buf, *d_buf;
    hipHostAlloc((void**)&h_buf, sizeof(int64_t) * thread_cnt,
	hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&d_buf, (void*)h_buf, 0);
    cudaCheckErrors("hipHostAlloc fail");
    for(int t = 0; t < thread_cnt; ++t) {
	h_buf[t] = -1;
    }

    // stream setup
    hipStream_t streamk;
    hipStreamCreate(&streamk);
    cudaCheckErrors("hipStreamCreate fail");

    // kernel call
    hipEvent_t eventk;
    hipEventCreateWithFlags(&eventk, hipEventDisableTiming);
    findOriginStructures<<<grid_size, block_size, 0, streamk>>>(start_seed,
	seed_range, d_buf);
    cudaCheckErrors("kernel launch fail");
    hipEventRecord(eventk, streamk);

    // writing buffer contents to output
    while(hipEventQuery(eventk) == hipErrorNotReady) {
	for(int t = 0; t < thread_cnt; ++t) {
	    if(h_buf[t] != -1) {
		int64_t seed = h_buf[t];
		printf("%lld\n", seed);
		fprintf(quadhut_list, "%lld\n", seed);
		fflush(quadhut_list);

		h_buf[t] = -1;
	    }
	}
    }
    // clean out buffer after kernel finishes
    for(int t = 0; t < thread_cnt; ++t) {
	if(h_buf[t] != -1) {
	    int64_t seed = h_buf[t];
	    printf("%lld\n", seed);
	    fprintf(quadhut_list, "%lld\n", seed);
	    fflush(quadhut_list);

	    h_buf[t] = -1;
	}
    }

    hipFree((void*)h_buf);

    return 0;
}
